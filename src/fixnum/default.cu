#include "hip/hip_runtime.h"
#pragma once

#include <math.h>
#include <type_traits>

#include "util/primitives.cu"
#include "slot_layout.cu"

/*
 * This is an archetypal implementation of a fixnum instruction
 * set. It defines the de facto interface for such implementations.
 *
 * All methods are defined for the device. It is someone else's
 * problem to get the data onto the device.
 */
template< int FIXNUM_BYTES_, typename word_tp_ = uint32_t >
class default_fixnum_impl {
    static_assert(FIXNUM_BYTES_ > 0,
            "Fixnum bytes must be positive.");
    static_assert(FIXNUM_BYTES_ % sizeof(word_tp_) == 0,
            "Fixnum word size must divide fixnum bytes.");
    static_assert(std::is_integral< word_tp_ >::value,
            "word_tp must be integral.");
    static constexpr int WORD_BITS = 8 * sizeof(word_tp_);

public:
    typedef word_tp_ word_tp;
    static constexpr int FIXNUM_BYTES = FIXNUM_BYTES_;
    static constexpr int SLOT_WIDTH = FIXNUM_BYTES_ / sizeof(word_tp_);
    // FIXME: slot_layout should not be exposed by this interface.
    typedef slot_layout< SLOT_WIDTH > slot_layout;
    typedef word_tp fixnum;

    /***************************
     * Representation functions.
     */

    /*
     * Set r using bytes, interpreting bytes as a base-256 unsigned
     * integer. Return the number of bytes used. If nbytes >
     * FIXNUM_BYTES, then the last nbytes - FIXNUM_BYTES are ignored.
     *
     * NB: Normally we would expect from_bytes to be exclusively a
     * device function, but it's the same for the host, so we leave it
     * in.
     */
    __host__ __device__ static int from_bytes(fixnum *r, const uint8_t *bytes, int nbytes) {
        uint8_t *s = reinterpret_cast< uint8_t * >(r);
        int n = min(nbytes, FIXNUM_BYTES);
        memcpy(s, bytes, n);
        memset(s + n, 0, FIXNUM_BYTES - n);
        return n;
    }

    /*
     * Set bytes using r, converting r to a base-256 unsigned
     * integer. Return the number of bytes written. If nbytes <
     * FIXNUM_BYTES, then the last FIXNUM_BYTES - nbytes are ignored.
     *
     * NB: Normally we would expect from_bytes to be exclusively a
     * device function, but it's the same for the host, so we leave it
     * in.
     */
    __host__ __device__ static int to_bytes(uint8_t *bytes, int nbytes, const fixnum *r) {
        int n = min(nbytes, FIXNUM_BYTES);
        memcpy(bytes, r, n);
        return n;
    }

    /*
     * load/set the value from ptr corresponding to this thread (lane) in
     * slot number idx.
     */
    __device__ static fixnum &load(fixnum *ptr, int idx = 0) {
        int off = idx * slot_layout::WIDTH + slot_layout::laneIdx();
        return ptr[off];
    }

    /*
     * Return digit at index idx.
     *
     * FIXME: Not clear how to interpret this function with more exotic fixnum
     * implementations such as RNS.
     */
    __device__ static fixnum get(fixnum var, int idx) {
        return slot_layout::shfl(var, idx);
    }

    /*
     * Return most significant digit.
     *
     * FIXME: Not clear how to interpret this function with more exotic fixnum
     * implementations such as RNS.
     */
    __device__ static fixnum most_sig_dig(fixnum var) {
        return slot_layout::shfl(var, slot_layout::toplaneIdx);
    }

    /***********************
     * Arithmetic functions.
     */

    // TODO: Handle carry in
    __device__ static int add_cy(fixnum &r, fixnum a, fixnum b) {
        int cy;
        r = a + b;
        cy = r < a;
        return resolve_carries(r, cy);
    }

    // TODO: Handle borrow in
    __device__ static int sub_br(fixnum &r, fixnum a, fixnum b) {
        int br;
        r = a - b;
        br = r > a;
        return resolve_borrows(r, br);
    }

    __device__ static fixnum zero() {
        return 0;
    }

    __device__ static fixnum one() {
        return (slot_layout::laneIdx() == 0);
    }

    __device__ static int incr_cy(fixnum &r) {
        return add_cy(r, r, one());
    }

    __device__ static int decr_br(fixnum &r) {
        return sub_br(r, r, one());
    }


    /*
     * r = lo_half(a * b)
     *
     * The "lo_half" is the product modulo 2^(8*FIXNUM_BYTES),
     * i.e. the same size as the inputs.
     */
    __device__ static void mul_lo(fixnum &r, fixnum a, fixnum b) {
        // TODO: This should be smaller, probably uint16_t (smallest
        // possible for addition).  Strangely, the naive translation to
        // the smaller size broke; to investigate.
        fixnum cy = 0;

        r = 0;
        for (int i = slot_layout::WIDTH - 1; i >= 0; --i) {
            fixnum aa = slot_layout::shfl(a, i);

            // TODO: See if using umad.wide improves this.
            umad_hi_cc(r, cy, aa, b, r);
            // TODO: Could use rotate here, which is slightly
            // cheaper than shfl_up0...
            r = slot_layout::shfl_up0(r, 1);
            cy = slot_layout::shfl_up0(cy, 1);
            umad_lo_cc(r, cy, aa, b, r);
        }
        cy = slot_layout::shfl_up0(cy, 1);
        add_cy(r, r, cy);
    }

    /*
     * (s, r) = a * b
     *
     * r is the "lo half" (see mul_lo above) and s is the
     * corresponding "hi half".
     */
    __device__ static void mul_wide(fixnum &s, fixnum &r, fixnum a, fixnum b) {
        // TODO: See if we can get away with a smaller type for cy.
        fixnum cy = 0;
        int L = slot_layout::laneIdx();

        // TODO: Rewrite this using rotates instead of shuffles;
        // should be simpler and faster.
        r = s = 0;
        for (int i = slot_layout::WIDTH - 1; i >= 0; --i) {
            fixnum aa = slot_layout::shfl(a, i), t;

            // TODO: Review this code: it seems to have more shuffles than
            // necessary, and besides, why does it not use digit_addmuli?
            umad_hi_cc(r, cy, aa, b, r);

            t = slot_layout::shfl(cy, slot_layout::toplaneIdx);
            // TODO: Is there a way to avoid this add?  Definitely need to
            // propagate the carry at least one place, but maybe not more?
            // Previous (wrong) version: "s = (L == 0) ? s + t : s;"
            t = (L == 0) ? t : 0;
            add_cy(s, s, t);

            // shuffle up hi words
            s = slot_layout::shfl_up(s, 1);
            // most sig word of lo words becomes least sig of hi words
            t = slot_layout::shfl(r, slot_layout::toplaneIdx);
            s = (L == 0) ? t : s;

            r = slot_layout::shfl_up0(r, 1);
            cy = slot_layout::shfl_up0(cy, 1);
            umad_lo_cc(r, cy, aa, b, r);
        }
        // TODO: This carry propgation from r to s is a bit long-winded.
        // Can we simplify?
        // NB: cy_hi <= width.  TODO: Justify this explicitly.
        fixnum cy_hi = slot_layout::shfl(cy, slot_layout::toplaneIdx);
        cy = slot_layout::shfl_up0(cy, 1);
        cy = add_cy(r, r, cy);
        cy_hi += cy;  // Can't overflow since cy_hi <= width.
        assert(cy_hi >= cy);
        // TODO: Investigate: replacing the following two lines with
        // simply "s = (L == 0) ? s + cy_hi : s;" produces no detectible
        // errors. Can I prove that (MAX_UINT64 - s[0]) < width?
        cy = (L == 0) ? cy_hi : 0;
        cy = add_cy(s, s, cy);
        assert(cy == 0);
    }

    /*
     * Return a mask of width bits whose ith bit is set if and only if
     * the ith digit of r is nonzero. In particular, result is zero
     * iff r is zero.
     */
    __device__ static uint32_t nonzero_mask(fixnum r) {
        return slot_layout::ballot(r != 0);
    }

    /*
     * Return -1, 0, or 1, depending on whether x is less than, equal
     * to, or greater than y.
     */
    __device__ static int cmp(fixnum x, fixnum y) {
        fixnum r;
        int br = sub_br(r, x, y);
        // r != 0 iff x != y. If x != y, then br != 0 => x < y.
        return nonzero_mask(r) ? (br ? -1 : 1) : 0;
    }

    /*
     * Return the index of the most significant bit of x, or -1 if x is
     * zero.
     */
    __device__ static int msb(fixnum x) {
        // FIXME: Should be able to get this value from limits or numeric_limits
        // or whatever.
        enum { UINT32_BITS = 8 * sizeof(uint32_t) };
        static_assert(UINT32_BITS == 32, "uint32_t isn't 32 bits");

        uint32_t a = nonzero_mask(x);
        // b is the index of the first non-zero word, or -1 if x is zero.
        int b = UINT32_BITS - (clz(a) + 1);
        if (b < 0) return b;
        word_tp y = slot_layout::shfl(x, b);
        int c = clz(y);
        return WORD_BITS - (c + 1) + WORD_BITS * b;
    }

private:
    __device__ static int resolve_carries(fixnum &r, int cy) {
        // FIXME: Can't call std::numeric_limits<fixnum>::max() on device.
        //static constexpr fixnum FIXNUM_MAX = std::numeric_limits<fixnum>::max();
        static constexpr fixnum FIXNUM_MAX = ~(fixnum)0;
        static constexpr int WIDTH = slot_layout::WIDTH;
        int L = slot_layout::laneIdx();
        uint32_t allcarries, p, g;
        int cy_hi;

        g = slot_layout::ballot(cy);              // carry generate
        p = slot_layout::ballot(r == FIXNUM_MAX); // carry propagate
        allcarries = (p | g) + g;                 // propagate all carries
        // FIXME: Unify these two expressions to remove the conditional;
        // the simple expression is not correct when WIDTH != warpSize
        //cy_hi = allcarries < g;                   // detect final overflow
        cy_hi = (WIDTH == 32) ? (allcarries < g) : ((allcarries >> WIDTH) & 1);
        allcarries = (allcarries ^ p) | (g << 1); // get effective carries
        r += (allcarries >> L) & 1;

        // return highest carry
        return cy_hi;
    }

    __device__ static int resolve_borrows(fixnum &r, int cy) {
        // FIXME: Use std::numeric_limits<fixnum>::min
        static constexpr fixnum FIXNUM_MIN = 0;
        static constexpr int WIDTH = slot_layout::WIDTH;
        int L = slot_layout::laneIdx();
        uint32_t allcarries, p, g;
        int cy_hi;

        g = slot_layout::ballot(cy);              // carry generate
        p = slot_layout::ballot(r == FIXNUM_MIN); // carry propagate
        allcarries = (p | g) + g;                 // propagate all carries
        // FIXME: Unify these two expressions to remove the conditional;
        // the simple expression is not correct when WIDTH != warpSize
        //cy_hi = allcarries < g;                   // detect final overflow
        cy_hi = (WIDTH == 32) ? (allcarries < g) : ((allcarries >> WIDTH) & 1);
        allcarries = (allcarries ^ p) | (g << 1); // get effective carries
        r -= (allcarries >> L) & 1;

        // return highest carry
        return cy_hi;
    }
};
