#pragma once
#include <hip/hip_runtime.h>


#include <stdint.h>

/*
 * Low-level/primitive functions.
 */

// TODO: Understand circumstances in which I might want to make this
// "#define ASM asm __volatile__".
//#define ASM asm

// hi * 2^32 + lo = a * b
__device__ __forceinline__ void
umul(uint32_t &hi, uint32_t &lo, uint32_t a, uint32_t b) {
    // TODO: Measure performance difference between this and the
    // equivalent:
    //   mul.hi.u32 %0, %2, %3
    //   mul.lo.u32 %1, %2, %3
    asm ("{\n\t"
         " .reg .u64 tmp;\n\t"
         " mul.wide.u32 tmp, %2, %3;\n\t"
         " mov.b64 { %0, %1 }, tmp;\n\t"
         "}"
         : "=r"(hi), "=r"(lo)
         : "r"(a), "r"(b));
}

__device__ __forceinline__ void
umul(uint64_t &r, uint32_t a, uint32_t b) {
    asm ("mul.wide.u32 %0, %1, %2;"
         : "=l"(r)
         : "r"(a), "r"(b));
}

// hi * 2^64 + lo = a * b
__device__ __forceinline__ void
umul(uint64_t &hi, uint64_t &lo, uint64_t a, uint64_t b) {
    asm ("mul.hi.u64 %0, %2, %3;\n\t"
         "mul.lo.u64 %1, %2, %3;"
         : "=l"(hi), "=l"(lo)
         : "l"(a), "l"(b));
}

// r = a * b + c
__device__ __forceinline__ void
umad(uint64_t &r, uint32_t a, uint32_t b, uint32_t c) {
    asm ("mad.wide.u32 %0, %1, %2, %3;"
         : "=l"(r)
         : "r"(a), "r"(b), "r"(c));
}

// (hi, lo) = a * b + c
__device__ __forceinline__ void
umad(uint32_t &hi, uint32_t &lo, uint32_t a, uint32_t b, uint32_t c) {
    asm ("{\n\t"
         " .reg .u64 tmp;\n\t"
         " mad.wide.u32 tmp, %2, %3, %4;\n\t"
         " mov.b64 { %0, %1 }, tmp;\n\t"
         "}"
         : "=r"(hi), "=r"(lo)
         : "r"(a), "r"(b), "r"(c));
}

// (hi, lo) = a * b + c
__device__ __forceinline__ void
umad(uint64_t &hi, uint64_t &lo, uint64_t a, uint64_t b, uint64_t c) {
    asm ("mad.lo.cc.u64 %1, %2, %3, %4;\n\t"
         "madc.hi.u64 %0, %2, %3, 0;"
         : "=l"(hi), "=l"(lo)
         : "l"(a), "l" (b), "l"(c));
}

// lo = a * b + c (mod 2^32)
__device__ __forceinline__ void
umad_lo(uint32_t &lo, uint32_t a, uint32_t b, uint32_t c) {
    asm ("mad.lo.u32 %0, %1, %2, %3;"
         : "=r"(lo)
         : "r"(a), "r" (b), "r"(c));
}

__device__ __forceinline__ void
umad_hi(uint32_t &hi, uint32_t a, uint32_t b, uint32_t c) {
    asm ("mad.hi.u32 %0, %1, %2, %3;"
         : "=r"(hi)
         : "r"(a), "r" (b), "r"(c));
}

// lo = a * b + c (mod 2^64)
__device__ __forceinline__ void
umad_lo(uint64_t &lo, uint64_t a, uint64_t b, uint64_t c) {
    asm ("mad.lo.u64 %0, %1, %2, %3;"
         : "=l"(lo)
         : "l"(a), "l" (b), "l"(c));
}

__device__ __forceinline__ void
umad_hi(uint64_t &hi, uint64_t a, uint64_t b, uint64_t c) {
    asm ("mad.hi.u64 %0, %1, %2, %3;"
         : "=l"(hi)
         : "l"(a), "l" (b), "l"(c));
}

// as above but with carry in cy
__device__ __forceinline__ void
umad_lo_cc(uint32_t &lo, uint32_t &cy, uint32_t a, uint32_t b, uint32_t c) {
    asm ("mad.lo.cc.u32 %0, %2, %3, %4;\n\t"
         "addc.u32 %1, %1, 0;"
         : "=r"(lo), "+r"(cy)
         : "r"(a), "r" (b), "r"(c));
}

__device__ __forceinline__ void
umad_hi_cc(uint32_t &lo, uint32_t &cy, uint32_t a, uint32_t b, uint32_t c) {
    asm ("mad.hi.cc.u32 %0, %2, %3, %4;\n\t"
         "addc.u32 %1, %1, 0;"
         : "=r"(lo), "+r"(cy)
         : "r"(a), "r" (b), "r"(c));
}

__device__ __forceinline__ void
umad_lo_cc(uint64_t &lo, uint64_t &cy, uint64_t a, uint64_t b, uint64_t c) {
    asm ("mad.lo.cc.u64 %0, %2, %3, %4;\n\t"
         "addc.u64 %1, %1, 0;"
         : "=l"(lo), "+l"(cy)
         : "l"(a), "l" (b), "l"(c));
}

__device__ __forceinline__ void
umad_hi_cc(uint64_t &lo, uint64_t &cy, uint64_t a, uint64_t b, uint64_t c) {
    asm ("mad.hi.cc.u64 %0, %2, %3, %4;\n\t"
         "addc.u64 %1, %1, 0;"
         : "=l"(lo), "+l"(cy)
         : "l"(a), "l" (b), "l"(c));
}


/*
 * Count Leading Zeroes in x.
 *
 * Use __builtin_clz{,l,ll}(x) or CUDA ASM depending on context.
 */
__host__ __device__ __forceinline__
int
clz(uint32_t x) {
#ifdef __CUDA_ARCH__
    int n;
    asm ("clz.b32 %0, %1;" : "=r"(n) : "r"(x));
    return n;
#else
    static_assert(sizeof(unsigned int) == sizeof(uint32_t),
            "attempted to use wrong __builtin_clz{,l,ll}()");
    return __builtin_clz(x);
#endif
}

__host__ __device__ __forceinline__
int
clz(uint64_t x) {
#ifdef __CUDA_ARCH__
    int n;
    asm ("clz.b64 %0, %1;" : "=r"(n) : "l"(x));
    return n;
#else
    static_assert(sizeof(unsigned long) == sizeof(uint64_t),
            "attempted to use wrong __builtin_clz{,l,ll}()");
    return __builtin_clzl(x);
#endif
}

/*
 * Return 1 if x = 2^n for some n, 0 otherwise.
 */
__host__ __device__ __forceinline__
int
is_binary_power(uint32_t x) {
    return ! (x & (x - 1));
}


/*
 * y >= x such that y = 2^n for some n. NB: This really is "inclusive"
 * next, i.e. if x is a binary power we just return it.
 */
__host__ __device__ __forceinline__
uint32_t
next_binary_power(uint32_t x) {
    enum { UINT32_BITS = 32 };
    return is_binary_power(x) ? x : (1 << (UINT32_BITS - clz(x)));
}


/*
 * ceiling(n / d) for integers.
 */
__host__ __device__ __forceinline__
int
iceil(int n, int d) {
    return (n + d - 1) / d;
}
